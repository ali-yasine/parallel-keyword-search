#include "hip/hip_runtime.h"
#include <vector>
#include <string>
#include <unordered_map>
#include <cmath>
#include <algorithm>
#include <string>
#include <cstring>
#include <random>
#include <cctype>  
#include <limits>
#include "util.h"

#define INF 2147483647

bool isSubstring(std::string haystack, std::string needle) {
    return haystack.find(needle) != std::string::npos;
}

void getQueryVertices(const std::vector<std::string>& query,
                      const std::unordered_map<int, std::string>& node_map,
                      const int num_nodes,
                      std::vector<std::unordered_set<int>>& query_vertices) {
    
    for (std::size_t i = 0; i < query.size(); ++i) {
        const auto& term = query[i];
        for (const auto& node_pair : node_map) {
            if (node_pair.first >= 0 && node_pair.first < num_nodes && isSubstring(node_pair.second, term)) {
                query_vertices[i].insert(node_pair.first);
            }
        }
    }
}

int getActivationLevel(const float node_weight, const float alpha, const  float avg_hops) {

    float epsilon = 0.00001; //hyperparameter try tweaking later

    //check if node_weight < alpha
    if (std::abs(node_weight - alpha) < epsilon) {
        return (int) std::round(avg_hops);
    }

    if (node_weight < alpha) {
        float reward = avg_hops * (alpha - node_weight) / alpha;
        return (int) std::round(avg_hops - reward);
    }

    float penalty = avg_hops * (node_weight - alpha) / (1 - alpha); 

    return (int) std::round(avg_hops + penalty);
}

void identify_central(const int num_nodes, bool* C_identifier, const bool* F_identifier , const int* M, const int query_num) {
    
    for (int node = 0; node < num_nodes; ++node) {

        if (F_identifier[node]) {
            bool is_central = true;
            for (int i = 0; i < query_num; ++i) {
                if (M[node * query_num + i] == INF) {
                    is_central = false;
                    break;
                }
            }
            if (is_central)
                C_identifier[node] = true;
        }
    }
}

void enqueue_frontier(const int num_nodes, bool* F_identifier, bool* frontier, int& frontier_size) {
    
    for (int i = 0; i < num_nodes; ++i) {
        frontier[i] = F_identifier[i];        
        F_identifier[i] = false;
    }
    frontier_size += num_nodes;
}

bool is_keyword(int node, const std::vector<std::unordered_set<int>>& keyword_nodes) {
    for (std::size_t i = 0; i < keyword_nodes.size(); ++i) {
        if (keyword_nodes[i].find(node) != keyword_nodes[i].end()) {
            return true;
        }
    }
    return false;
}

void getMinActivations(const float* node_weights, const int num_nodes, const float alpha, const float avg_hops, int* min_activations) {
    for (int i = 0; i < num_nodes; ++i) {
        min_activations[i] = getActivationLevel(node_weights[i], alpha, avg_hops);
    }
}

float score(const CooGraph* graph, const int central_node, const float* node_weights, const int* M, const int query_num, const float lambda) {

    int depth = 0;
    for (int i = 0; i < query_num; ++i) {
        if (M[central_node * query_num + i] > depth) {
            depth = M[central_node * query_num + i];
        }
    }

    float total_weight = 0;
    std::unordered_set<int> visitedSet {};
    
    for(int edge = 0; edge < graph->num_edges; ++edge) {
        int node = graph->col_indices[edge];

        if (visitedSet.find(node) == visitedSet.end()) {
            total_weight += node_weights[node];
            visitedSet.insert(node);
        }
    }
    
    return powf((float) depth, lambda) * total_weight;
}

void dequeue_frontier(bool* frontier, int& frontier_size, const int num_nodes) {
    for (int i = 0; i < num_nodes; ++i) {
        frontier[i] = 0;
    }
    frontier_size = 0;
}

bool check_terminate(const bool* C_identifier, const int num_nodes, const int k) {
    
    int count = 0;
    
    for (int i = 0; i < num_nodes; ++i) {
        if (C_identifier[i])
            count++;
    }

    return count >= k;

}


#undef INF