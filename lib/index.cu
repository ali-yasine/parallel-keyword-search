#include "hip/hip_runtime.h"
#include <unordered_map>
#include <unordered_set>
#include <vector>
#include <cmath>
#include <queue>
#include <algorithm>
#include <iostream>
#include <fstream>

#include "graph.h"
#include "index.h"
#include "util.h"

using std::vector;  
using std::unordered_map;

struct pair_hash {
    template <class T1, class T2>
    std::size_t operator () (const std::pair<T1, T2>& p) const {
        auto h1 = std::hash<T1>{}(p.first);
        auto h2 = std::hash<T2>{}(p.second);
        return h1 ^ h2;
    }
};

struct pair_equal {
    template <class T1, class T2>
    bool operator () (const std::pair<T1, T2>& lhs, const std::pair<T1, T2>& rhs) const {
        return lhs.first == rhs.first && lhs.second == rhs.second;
    }
};

void getVertexInformativeness(const CsrGraph* csr, const CooGraph* coo,  float* vertex_w, float& avg_hops) {
    std::cerr << "Calculating vertex informativeness...\n";
    float max_w = std::numeric_limits<float>::min();
    float min_w = std::numeric_limits<float>::max();

    avg_hops = averageDistanceInGraph(csr);

    std::vector<std::unordered_map<int, int>> incoming_label_counts(csr->num_nodes, unordered_map<int, int> {});
    
    for(int edge = 0; edge < coo->num_edges; ++edge) {
        int dst = coo->col_indices[edge];
        int label = coo->edge_labels[edge];
        ++incoming_label_counts[dst][label];
    }

    for (int node = 0; node < csr->num_nodes; ++node) {
        const auto& label_counts = incoming_label_counts[node];
        int total_count = 0;
        float informativeness = 0.0f;

        for (const auto& [label, count] : label_counts) {
            total_count += count;
            informativeness += count * std::log2(1.0f + count);
        }
        if (total_count > 0) {
            informativeness /= total_count;
        }
        else {
            informativeness = 0.0f;
        }
        vertex_w[node] = informativeness;

        //update max and min
        min_w = std::min(min_w, informativeness);
        max_w = std::max(max_w, informativeness);
    }
    float range = max_w - min_w;
    for (int node = 0; node < csr->num_nodes; ++node) {
        vertex_w[node] = (vertex_w[node] - min_w) / range;
    }
}

int bfs(const CsrGraph* graph, const int src, const int dst) {
    std::vector<bool> visited(graph->num_nodes, false);
    std::queue<std::pair<int, int>> q {};
    q.push({src, 0});
    visited[src] = true;

    while (!q.empty()) {
        auto [node, distance] = q.front();
        q.pop();
        if (node == dst) {
            return distance;
        }
        
        for (int edge = graph->row_offsets[node]; edge < graph->row_offsets[node + 1]; ++edge) {
            int neighbor = graph->col_indices[edge];
            if (!visited[neighbor]) {
                q.push({neighbor, distance + 1});
                visited[neighbor] = true;
            }
        }
    }

    return -1;
}
float averageDistanceInGraph(const CsrGraph* graph, int num_samples) {
    
    //get average number of hops needed to reach a vertex from another vertex by sampling
    int n = graph->num_nodes;
    RandomGen rand_gen {0, n - 1};
    int total_distance = 0;

    for(int i = 0; i < num_samples; ++i) {
        
        int src = rand_gen();
        int dst = rand_gen();
        int distance = bfs(graph, src, dst);

        if (distance == -1) {
            --i;
            continue;
        }

        total_distance += distance;
    }

    return static_cast<float>(total_distance) / num_samples;
}

void writeGraphIndex(const char* filename, float* vertex_w, float avg_hops, int graph_size) {
    FILE* fp = fopen(filename, "w");
    fprintf(fp, "%lf\n", avg_hops);
    
    for(int i = 0; i < graph_size; ++i) {
        fprintf(fp, "%d: %f \n", i, vertex_w[i]);
    }

    fclose(fp);
}

void readGraphIndex(const char* filename, float* vertex_w, float& avg_hops, int graph_size) {
    FILE* fp = fopen(filename, "r");
    fscanf(fp, "%f\n", &avg_hops);
    int temp;
    for(int i = 0; i < graph_size; ++i) {
        fscanf(fp, "%d: %f ", &temp, &vertex_w[i]);
    }
    fclose(fp);
}
