#include "hip/hip_runtime.h"
#include <unordered_set>
#include <vector>
#include <cassert>
#include <cmath>
#include <iostream>

#include "gpu_util.h"
#include "graph.h"

#define MAX_THREADS 1024
#define INF 2147483647
#define ull unsigned long long

CsrGraph* createEmptyCsrGPU(int num_nodes, int num_edges) {
    CsrGraph graph_shadow;
    graph_shadow.num_nodes = num_nodes;
    graph_shadow.num_edges = num_edges;

    hipMalloc((void**)&graph_shadow.row_offsets, (num_nodes + 1) * sizeof(int));
    hipMalloc((void**)&graph_shadow.col_indices, num_edges * sizeof(int));
    hipMalloc((void**)&graph_shadow.edge_labels, num_edges * sizeof(int));

    CsrGraph* graph;
    hipMalloc((void**)&graph, sizeof(CsrGraph));
    hipMemcpy(graph, &graph_shadow, sizeof(CsrGraph), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    return graph;
}

void copyCsrGraphToDevice(const CsrGraph* graph, CsrGraph* graph_d) {
    CsrGraph graph_shadow;

    hipMemcpy(&graph_shadow, graph_d, sizeof(CsrGraph), hipMemcpyDeviceToHost);
    assert(graph_shadow.num_nodes == graph->num_nodes);
    assert(graph_shadow.num_edges == graph->num_edges);

    hipMemcpy(graph_shadow.row_offsets, graph->row_offsets, (graph->num_nodes + 1) * sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(graph_shadow.col_indices, graph->col_indices, graph->num_edges * sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(graph_shadow.edge_labels, graph->edge_labels, graph->num_edges * sizeof(int), hipMemcpyHostToDevice);
}


void freeCsrGPU(CsrGraph* graph) {
    hipFree(graph->row_offsets);
    hipFree(graph->col_indices);
    hipFree(graph->edge_labels);
    hipFree(graph);
}


__global__ void countTruesKernel(const bool* array, int size, int* count){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size)
    {
        if (array[tid])
        {
            atomicAdd(count, 1);
        }
    }
}

int countOnesGPU(const bool* array, int size)
{
    // Allocate memory on the GPU for the array and count variables
    int* count_d;
    hipMalloc((void**)&count_d, sizeof(int));

    // Initialize the count to 0
    auto err = hipMemset(count_d, 0, sizeof(int));

    if (err != hipSuccess)
    {
        // Handle hipMemset error
        std::cerr << "Error during hipMemset countOnes: " << hipGetErrorString(err) << "\n";
    }

    // Launch the CUDA kernel
    int threadsPerBlock = MAX_THREADS;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    countTruesKernel<<<blocksPerGrid, threadsPerBlock>>>(array, size, count_d);

    hipDeviceSynchronize();

    // Copy the result (count) back to the host
    int count;
    hipMemcpy(&count, count_d, sizeof(int), hipMemcpyDeviceToHost);

    // Free the allocated memory on the GPU
    hipFree(count_d);

    return count;
}



void init_keyword_nodes_and_M_onGPU(bool* keyword_nodes_d, int* M_d, int num_nodes, int query_num, const std::vector<std::unordered_set<int>>& keyword_nodes) {

    hipError_t err;
    bool* keyword_nodes_h = (bool*) calloc(num_nodes, sizeof(bool));

    int* M_h = (int*) malloc(num_nodes * query_num * sizeof(int));
    
    for(int i = 0; i < num_nodes * query_num; i++) {
        M_h[i] = INF;
    }
    

    for (int i = 0; i < query_num; i++) {
        for (auto node : keyword_nodes[i]) {
            keyword_nodes_h[node] = true;
            M_h[node * query_num + i] = 0;
        }
    }

    //count number of keyword nodes
    int num_keyword_nodes = 0;

    for (int i = 0; i < num_nodes; i++) {
        if (keyword_nodes_h[i]) {
            num_keyword_nodes++;
        }
    }


    hipMemcpy(keyword_nodes_d, keyword_nodes_h, num_nodes * sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(M_d, M_h, num_nodes * query_num * sizeof(int), hipMemcpyHostToDevice);
    err = hipGetLastError();
    if (err != hipSuccess) {
        // Handle hipMemcpy error
        std::cerr << "Error during hipMemcpy: " << hipGetErrorString(err) << std::endl;
        // Clean up allocated memory
        free(M_h);
        free(keyword_nodes_h);
        return;
    }

    hipDeviceSynchronize();
    
    free(M_h);
    free(keyword_nodes_h);
}


void init_M_keywords_bitwise(ull* keyword_nodes_d, int* M_d, int num_nodes, int query_num, const std::vector<std::unordered_set<int>>& keyword_nodes) {


    hipError_t err;
    int num_nodes_ull = (num_nodes + 64 - 1) / 64;

    ull* keyword_nodes_h = (ull*) calloc(num_nodes_ull, sizeof(ull));

    int* M_h = (int*) malloc(num_nodes * query_num * sizeof(int));

    for(int i = 0; i < num_nodes * query_num; i++) {
        M_h[i] = INF;
    }

    for (int i = 0; i < query_num; i++) {
        for (auto node : keyword_nodes[i]) {
            keyword_nodes_h[node / 64] |= (1ULL << (node % 64));
            M_h[node * query_num + i] = 0;
        }
    }

    hipMemcpy(keyword_nodes_d, keyword_nodes_h, num_nodes_ull * sizeof(ull), hipMemcpyHostToDevice);
    hipMemcpy(M_d, M_h, num_nodes * query_num * sizeof(int), hipMemcpyHostToDevice);
    err = hipGetLastError();

    if (err != hipSuccess) {
        // Handle hipMemcpy error
        std::cerr << "Error during hipMemcpy: " << hipGetErrorString(err) << std::endl;
        // Clean up allocated memory
        free(M_h);
        free(keyword_nodes_h);
        return;
    }

    hipDeviceSynchronize();
    free(M_h);
    free(keyword_nodes_h);

}

__device__ int getActivationLevel_gpu(const float node_weight, float alpha, float avg_hops) {

    float epsilon = 0.0001f;

    //check if node_weight = alpha
    if (fabsf(node_weight - alpha) < epsilon) {
        return (int) roundf(avg_hops);
    }

    if (node_weight < alpha) {
        float reward = avg_hops * (alpha - node_weight) / alpha;
        return (int) roundf(avg_hops - reward);
    }

    float penalty = avg_hops * (node_weight - alpha) / (1.0f - alpha); 

    return (int) roundf(avg_hops + penalty);

}

__global__ void get_min_activations_kernel(const float* node_weights, int num_nodes, float alpha, int avg_hops, int* min_activations) {

    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id < num_nodes) {
        min_activations[thread_id] = getActivationLevel_gpu(node_weights[thread_id], alpha, avg_hops);
    }
}

void get_min_activations_gpu(const float* node_weights, const int num_nodes, const float alpha, const float avg_hops, int* min_activations_d) {
    
    const unsigned int num_threads = MAX_THREADS;
    const unsigned int num_blocks = ((num_nodes) + num_threads - 1) / num_threads;

    float* node_weights_d;
    hipMalloc((void**)&node_weights_d, num_nodes * sizeof(float));
    hipMemcpy(node_weights_d, node_weights, num_nodes * sizeof(float), hipMemcpyHostToDevice);

    get_min_activations_kernel <<<num_blocks, num_threads >>> (node_weights_d, num_nodes, alpha, avg_hops, min_activations_d);

    hipFree(node_weights_d);
}


void enqueue_frontier_gpu(int num_nodes, bool* F_identifier_d, bool* frontier_d) {
    //copy F_identifier_d to frontier_d
    
    auto err = hipMemcpy(frontier_d, F_identifier_d, num_nodes * sizeof(bool), hipMemcpyDeviceToDevice);
    
    if (err != hipSuccess) {
        // Handle hipMemcpy error
        std::cerr << "Error during hipMemcpy enque_frontier: " << hipGetErrorString(err) << "\n";
    }

    err = hipMemset(F_identifier_d, 0, num_nodes * sizeof(bool));

    if (err != hipSuccess) {
        // Handle hipMemset error
        std::cerr << "Error during hipMemset enque_frontier: " << hipGetErrorString(err) << "\n";
    }
    
}

void enqueue_frontier_bitwise(int num_nodes, ull* F_identifier_d, ull* frontier_d) {

    int num_nodes_ull = (num_nodes + 64 - 1) / 64;

    auto err = hipMemcpy(frontier_d, F_identifier_d, num_nodes_ull * sizeof(ull), hipMemcpyDeviceToDevice);

    if (err != hipSuccess) {
        // Handle hipMemcpy error
        std::cerr << "Error during hipMemcpy enque_frontier: " << hipGetErrorString(err) << "\n";
    }

    err = hipMemset(F_identifier_d, 0, num_nodes_ull * sizeof(ull));

    if (err != hipSuccess) {
        // Handle hipMemset error
        std::cerr << "Error during hipMemset enque_frontier: " << hipGetErrorString(err) << "\n";
    }

}

__global__ void identify_central_kernel(int num_nodes, bool* C_identifier, bool* F_identifier, int* M, int query_num) {
    
    int node = blockIdx.x * blockDim.x + threadIdx.x;
    if (node < num_nodes) {
        if (F_identifier[node]) {
            bool is_central = true;
            for (int i = 0; i < query_num; i++) {
                if (M[node * query_num + i] == INF) {
                    is_central = false;
                    break;
                }
            }
            if (is_central) {
                C_identifier[node] = true;
            }
        }
    }
}


void identify_central_gpu(int num_nodes, bool* C_identifier_d, bool* F_identifier_d, int* M_d, int query_num) {
    const unsigned int num_threads = MAX_THREADS;
    const unsigned int num_blocks = ((num_nodes) + num_threads - 1) / num_threads;

    identify_central_kernel << <num_blocks, num_threads >> > (num_nodes, C_identifier_d, F_identifier_d, M_d, query_num);

    hipDeviceSynchronize();   
}


__global__ void identify_central_bitwise_kernel(int num_nodes, ull* C_identifier, ull* F_identifier, int* M, int query_num) {

    int node = blockIdx.x * blockDim.x + threadIdx.x;
    if (node < num_nodes) {
        if (F_identifier[node / 64] & (1ULL << (node % 64))) {
            bool is_central = true;
            for (int i = 0; i < query_num; i++) {
                if (M[node * query_num + i] == INF) {
                    is_central = false;
                    break;
                }
            }
            if (is_central) {
                atomicOr(&C_identifier[node / 64], 1ULL << (node % 64));
            }
        }
    }
}


void identify_central_bitwise(int num_nodes, ull* C_identifier_d, ull* F_identifier_d, int* M_d, int query_num) {
    const unsigned int num_threads = MAX_THREADS;
    const unsigned int num_blocks = ((num_nodes) + num_threads - 1) / num_threads;

    identify_central_bitwise_kernel << <num_blocks, num_threads >> > (num_nodes, C_identifier_d, F_identifier_d, M_d, query_num);

    hipDeviceSynchronize();
}


void dequeue_frontier_gpu(bool* frontier_d, int num_nodes) {
    hipMemsetAsync(frontier_d, 0, num_nodes * sizeof(bool));
}

void dequeue_frontier_bitwise(ull* frontier_d, int num_nodes) {
    int num_nodes_ull = (num_nodes + 64 - 1) / 64;
    hipMemsetAsync(frontier_d, 0, num_nodes_ull * sizeof(ull));
}

bool check_terminate_gpu(bool* C_identifier_d, int num_nodes, int k) {


    int count = countOnesGPU(C_identifier_d, num_nodes);
            
    return count >= k;
}

__global__ void countOnesBitwiseGPU(ull* C_identifier, int num_nodes_ull, int* count) {
    int node = blockIdx.x * blockDim.x + threadIdx.x;
    if (node < num_nodes_ull) {

        int num_centrals = __popcll(C_identifier[node]);

        if (num_centrals > 0)
            atomicAdd(count, num_centrals);
    }
}


bool check_terminate_bitwise(ull* C_identifier_d, int num_nodes, int k) {

    int num_nodes_ull = (num_nodes + 64 - 1) / 64;

    int threadsPerBlock = MAX_THREADS;
    int numBlocks = (num_nodes_ull + threadsPerBlock - 1) / threadsPerBlock;

    int* count_d; 
    hipMalloc((void**)&count_d, sizeof(int));

    hipMemset(count_d, 0, sizeof(int));

    countOnesBitwiseGPU << <numBlocks, threadsPerBlock >> > (C_identifier_d, num_nodes_ull, count_d);

    int count;

    hipMemcpy(&count, count_d, sizeof(int), hipMemcpyDeviceToHost);


    hipFree(count_d);

    return count >= k;
}

void cudaFreeGraph(CsrGraph* graph_d) {
    hipFree(graph_d->row_offsets);
    hipFree(graph_d->col_indices);
    hipFree(graph_d->edge_labels);
    hipFree(graph_d);
}


__global__ void init_M_kernel(bool* is_keyword_d, int* M_d, int num_nodes, int query_num) {
    int node = blockIdx.x * blockDim.x + threadIdx.x;
    if (node < num_nodes) {
        if (is_keyword_d[node]) {
            for (int i = 0; i < query_num; i++) {
                M_d[node * query_num + i] = 0;
            }
        }
        else {
            for (int i = 0; i < query_num; i++) {
                M_d[node * query_num + i] = INF;
            }
        }
    }
}

void init_M_gpu(bool* is_keyword_d, int* M_d, int num_nodes, int query_num) {
    const unsigned int num_threads = MAX_THREADS;
    const unsigned int num_blocks = ((num_nodes) + num_threads - 1) / num_threads;

    init_M_kernel << <num_blocks, num_threads >> > (is_keyword_d, M_d, num_nodes, query_num);

    hipDeviceSynchronize();
}