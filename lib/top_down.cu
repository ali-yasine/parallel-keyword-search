#include "hip/hip_runtime.h"
#include <algorithm>
#include <deque>
#include <unordered_set>
#include <vector>
#include <queue>

#include <iostream>
#include "top_down.h"
#include "util.h"
#include "graph.h"
#define INF 2147483647
#define ull unsigned long long


struct Compare_graphs {
    bool operator()(const std::pair<CooGraph*, int>& p1, const std::pair<CooGraph*, int>& p2) {
        return p1.second < p2.second;
    }
};


void topdown_construct(const CsrGraph* graph, CooGraph** result, const bool* C_identifier, const int* M, const int query_num, const int k, const std::vector<std::unordered_set<int>> keyword_nodes, const int* min_activations, const float* node_weights) {
    
    int* costs = (int*) calloc(k, sizeof(int));
    int* is_next_frontier = (int*) malloc(graph->num_nodes * sizeof(int));

    std::priority_queue<std::pair<CooGraph*, int>, std::vector<std::pair<CooGraph*, int>>, Compare_graphs> heap {};

    for(int central_node = 0; central_node < graph->num_nodes; ++central_node) {
        
        if (C_identifier[central_node]) {

            memset(is_next_frontier, 0, graph->num_nodes * sizeof(int));
            
            CooGraph* curr_graph = (CooGraph*) malloc(sizeof(CooGraph));
            curr_graph->num_nodes = 1;
            curr_graph->num_edges = 0;
            curr_graph->row_indices = (int*) malloc(graph->num_edges * sizeof(int));
            curr_graph->col_indices = (int*) malloc(graph->num_edges * sizeof(int));
            curr_graph->edge_labels = (int*) malloc(graph->num_edges * sizeof(int));

            std::deque<int> frontier {}, next_frontier {};
            std::unordered_set<int> added_nodes {central_node};
            frontier.push_front(central_node);
            
            while (!frontier.empty()) {

                int curr_node = frontier.front();
                frontier.pop_front();
                next_frontier.clear();
                int min_activation_level = min_activations[curr_node];

                //scan the neighbors of curr_node
                for(int neighbor = graph->row_offsets[curr_node]; neighbor < graph->row_offsets[curr_node + 1]; ++neighbor) {
                    
                    int neighbor_id = graph->col_indices[neighbor];
                    int edge_label = graph->edge_labels[neighbor];
                    bool added = false;

                    for (int bfs_instance = 0; bfs_instance < query_num; ++bfs_instance) {

                        int curr_hitting_level = M[curr_node * query_num + bfs_instance];
                        int neighbor_hitting_level = M[neighbor_id * query_num + bfs_instance];
                        int neighbor_activation_level = min_activations[neighbor_id];

                        if (is_keyword(curr_node, keyword_nodes) && curr_hitting_level == 1 + std::max(neighbor_activation_level , neighbor_hitting_level)) {
                            
                            curr_graph->row_indices[curr_graph->num_edges] = curr_node;
                            curr_graph->col_indices[curr_graph->num_edges] = neighbor_id;
                            curr_graph->edge_labels[curr_graph->num_edges++] = edge_label;

                            curr_graph->row_indices[curr_graph->num_edges] = neighbor_id;
                            curr_graph->col_indices[curr_graph->num_edges] = curr_node;
                            curr_graph->edge_labels[curr_graph->num_edges++] = edge_label;
                            
                            //check if neighbor in the next frontier
                            if (!is_next_frontier[neighbor_id]) {
                                next_frontier.push_front(neighbor_id);
                                is_next_frontier[neighbor_id] = true;
                            }
                            added = true;
                        }

                        else {
                            if (curr_hitting_level == 1 + std::max({neighbor_activation_level, neighbor_hitting_level, min_activation_level - 1})) {
                            
                                curr_graph->row_indices[curr_graph->num_edges] = curr_node;
                                curr_graph->col_indices[curr_graph->num_edges] = neighbor_id;
                                curr_graph->edge_labels[curr_graph->num_edges++] = edge_label;

                                curr_graph->row_indices[curr_graph->num_edges] = neighbor_id;
                                curr_graph->col_indices[curr_graph->num_edges] = curr_node;
                                curr_graph->edge_labels[curr_graph->num_edges++] = edge_label;


                                if (!is_next_frontier[neighbor_id]) {
                                    
                                    is_next_frontier[neighbor_id] = true;
                                    next_frontier.push_back(neighbor_id);
                                }
                                added = true;
                            }
                        }

                        if (added) {
                            added_nodes.insert(neighbor_id);
                            frontier = next_frontier;
                            break;
                        }
                    }
                }   
            }
            
            curr_graph->row_indices = (int*) realloc(curr_graph->row_indices, curr_graph->num_edges * sizeof(int));
            curr_graph->col_indices = (int*) realloc(curr_graph->col_indices, curr_graph->num_edges * sizeof(int));
            curr_graph->edge_labels = (int*) realloc(curr_graph->edge_labels, curr_graph->num_edges * sizeof(int));
            curr_graph->num_nodes = added_nodes.size();
            
            if (!curr_graph->row_indices || !curr_graph->col_indices || !curr_graph->edge_labels) {
                std::cerr << "Error: Memory allocation failed realloc top_down\n";
                freeGraph(curr_graph);
                continue;
            }

            level_cover(curr_graph, keyword_nodes, central_node);

            int curr_score = score(curr_graph, central_node, node_weights, M, query_num);

            if (heap.size() < k) {
                heap.push(std::make_pair(curr_graph, curr_score));
            }
            else {
                if (curr_score < heap.top().second) {
                    freeGraph(heap.top().first);
                    heap.pop();
                    heap.push(std::make_pair(curr_graph, curr_score));
                }
                else {
                    freeGraph(curr_graph);
                }
            }
        }
    }
    free(is_next_frontier);

    for(int i = k - 1; i >= 0; --i) {
        result[i] = heap.top().first;
        heap.pop();
    }
    free(costs);
}


void topdown_construct(const CsrGraph* graph, CooGraph** result, const ull* C_identifier, const int* M, const int query_num, const int k, const std::vector<std::unordered_set<int>> keyword_nodes, const int* min_activations, const float* node_weights) {
    
    int* costs = (int*) calloc(k, sizeof(int));
    int* is_next_frontier = (int*) malloc(graph->num_nodes * sizeof(int));

    std::priority_queue<std::pair<CooGraph*, int>, std::vector<std::pair<CooGraph*, int>>, Compare_graphs> heap {};

    for(int central_node = 0; central_node < graph->num_nodes; ++central_node) {
        
        if (C_identifier[central_node / 64] & (1ULL << (central_node % 64))) {

            memset(is_next_frontier, 0, graph->num_nodes * sizeof(int));
            
            CooGraph* curr_graph = (CooGraph*) malloc(sizeof(CooGraph));
            curr_graph->num_nodes = 1;
            curr_graph->num_edges = 0;
            curr_graph->row_indices = (int*) malloc(graph->num_edges * sizeof(int));
            curr_graph->col_indices = (int*) malloc(graph->num_edges * sizeof(int));
            curr_graph->edge_labels = (int*) malloc(graph->num_edges * sizeof(int));

            std::deque<int> frontier {}, next_frontier {};
            std::unordered_set<int> added_nodes {central_node};
            frontier.push_front(central_node);
            
            while (!frontier.empty()) {

                int curr_node = frontier.front();
                frontier.pop_front();
                next_frontier.clear();
                int min_activation_level = min_activations[curr_node];

                //scan the neighbors of curr_node
                for(int neighbor = graph->row_offsets[curr_node]; neighbor < graph->row_offsets[curr_node + 1]; ++neighbor) {
                    
                    int neighbor_id = graph->col_indices[neighbor];
                    int edge_label = graph->edge_labels[neighbor];
                    bool added = false;

                    for (int bfs_instance = 0; bfs_instance < query_num; ++bfs_instance) {

                        int curr_hitting_level = M[curr_node * query_num + bfs_instance];
                        int neighbor_hitting_level = M[neighbor_id * query_num + bfs_instance];
                        int neighbor_activation_level = min_activations[neighbor_id];

                        if (is_keyword(curr_node, keyword_nodes) && curr_hitting_level == 1 + std::max(neighbor_activation_level , neighbor_hitting_level)) {
                            
                            curr_graph->row_indices[curr_graph->num_edges] = curr_node;
                            curr_graph->col_indices[curr_graph->num_edges] = neighbor_id;
                            curr_graph->edge_labels[curr_graph->num_edges++] = edge_label;

                            curr_graph->row_indices[curr_graph->num_edges] = neighbor_id;
                            curr_graph->col_indices[curr_graph->num_edges] = curr_node;
                            curr_graph->edge_labels[curr_graph->num_edges++] = edge_label;
                            
                            //check if neighbor in the next frontier
                            if (!is_next_frontier[neighbor_id]) {
                                next_frontier.push_front(neighbor_id);
                                is_next_frontier[neighbor_id] = true;
                            }
                            added = true;
                        }

                        else {
                            if (curr_hitting_level == 1 + std::max({neighbor_activation_level, neighbor_hitting_level, min_activation_level - 1})) {
                            
                                curr_graph->row_indices[curr_graph->num_edges] = curr_node;
                                curr_graph->col_indices[curr_graph->num_edges] = neighbor_id;
                                curr_graph->edge_labels[curr_graph->num_edges++] = edge_label;

                                curr_graph->row_indices[curr_graph->num_edges] = neighbor_id;
                                curr_graph->col_indices[curr_graph->num_edges] = curr_node;
                                curr_graph->edge_labels[curr_graph->num_edges++] = edge_label;


                                if (!is_next_frontier[neighbor_id]) {
                                    
                                    is_next_frontier[neighbor_id] = true;
                                    next_frontier.push_back(neighbor_id);
                                }
                                added = true;
                            }
                        }

                        if (added) {
                            added_nodes.insert(neighbor_id);
                            frontier = next_frontier;
                            break;
                        }
                    }
                }   
            }
            
            curr_graph->row_indices = (int*) realloc(curr_graph->row_indices, curr_graph->num_edges * sizeof(int));
            curr_graph->col_indices = (int*) realloc(curr_graph->col_indices, curr_graph->num_edges * sizeof(int));
            curr_graph->edge_labels = (int*) realloc(curr_graph->edge_labels, curr_graph->num_edges * sizeof(int));
            curr_graph->num_nodes = added_nodes.size();
            
            if (!curr_graph->row_indices || !curr_graph->col_indices || !curr_graph->edge_labels) {
                std::cerr << "Error: Memory allocation failed realloc top_down\n";
                freeGraph(curr_graph);
                continue;
            }

            level_cover(curr_graph, keyword_nodes, central_node);

            int curr_score = score(curr_graph, central_node, node_weights, M, query_num);

            if (heap.size() < k) {
                heap.push(std::make_pair(curr_graph, curr_score));
            }
            else {
                if (curr_score < heap.top().second) {
                    freeGraph(heap.top().first);
                    heap.pop();
                    heap.push(std::make_pair(curr_graph, curr_score));
                }
                else {
                    freeGraph(curr_graph);
                }
            }
        }
    }
    free(is_next_frontier);

    for(int i = k - 1; i >= 0; --i) {
        result[i] = heap.top().first;
        heap.pop();
    }
    free(costs);
}

void level_cover(CooGraph*& graph, const std::vector<std::unordered_set<int>> keyword_nodes, const int central_node) {
    std::unordered_map<int, int> keyword_count {};
    std::unordered_set<int> visited {};
    int max_count = 0;

    CooGraph* result = (CooGraph*) malloc(sizeof(CooGraph));
    result->row_indices = (int*) malloc(graph->num_edges * sizeof(int));
    result->col_indices = (int*) malloc(graph->num_edges * sizeof(int));
    result->edge_labels = (int*) malloc(graph->num_edges * sizeof(int));
    result->num_edges = 0;
    result->num_nodes = 0;

    for(int edge = 0; edge < graph->num_edges; ++edge) {
        int src = graph->row_indices[edge];
        if (visited.find(src) == visited.end()) {
            for(int keyword = 0; keyword < keyword_nodes.size(); ++keyword) {
                if (keyword_nodes[keyword].find(src) != keyword_nodes[keyword].end())  {
                    keyword_count[src]++;
                    if (keyword_count[src] > max_count) 
                        max_count = keyword_count[src];
                }
            }
            visited.insert(src);
        }
    }
    visited.clear();

    std::unordered_set<int> cover_nodes {};

    std::vector<std::unordered_set<int>> levels(max_count + 1, std::unordered_set<int> {});
    for (auto it = keyword_count.begin(); it != keyword_count.end(); ++it) {
        if (it->first != central_node) 
            levels[it->second].insert(it->first);
        else
            levels[max_count].insert(it->first);
    }

    bool* covered = (bool*) calloc(keyword_nodes.size(), sizeof(bool));

    for(int level = max_count; level > 0; --level) {
        for (auto it = levels[level].begin(); it != levels[level].end(); ++it) {
            cover_nodes.insert(*it);
            for(int keyword = 0; keyword < keyword_nodes.size(); ++keyword) {
                if (keyword_nodes[keyword].find(*it) != keyword_nodes[keyword].end()){
                    if (!covered[keyword]) {
                        covered[keyword] = true;
                        break;
                    }
                }
            }
        }
        
        //if all keywords are covered, stop
        if (std::all_of(covered, covered + keyword_nodes.size(), [](bool b) {return b;})) {
            break;
        }
    }
    
    free(covered);

    result->num_nodes = cover_nodes.size();
    //prune levels that are not needed
    for (int edge = 0; edge < graph->num_edges; ++edge) {
        int src = graph->row_indices[edge];
        int dst = graph->col_indices[edge];
        
        if (cover_nodes.find(src) != cover_nodes.end() && cover_nodes.find(dst) != cover_nodes.end()) {
            result->row_indices[result->num_edges] = src;
            result->col_indices[result->num_edges] = dst;
            result->edge_labels[result->num_edges++] = graph->edge_labels[edge];
        }
    }
    
        
    result->row_indices = (int*) realloc(result->row_indices, result->num_edges * sizeof(int));
    result->col_indices = (int*) realloc(result->col_indices, result->num_edges * sizeof(int));
    result->edge_labels = (int*) realloc(result->edge_labels, result->num_edges * sizeof(int));

    if (!result->row_indices || !result->col_indices || !result->edge_labels) {
        freeGraph(result);
        return;
    }

    freeGraph(graph);
    graph = result;
}

#undef INF