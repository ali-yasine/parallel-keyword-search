#include <iostream>
#include <unordered_map>
#include <string>
#include <cstring>
#include <fstream>
#include <unistd.h>
#include <sstream>
#include <vector>
#include <unordered_set>
#include <algorithm>

#include "graph.h"
#include "index.h"
#include "pks.h"
#include "util.h"
#include "timer.h"
#include "gpu_util.h"


int main(int argc, char** argv) {
    Timer timer;
    std::vector<std::string> query;
    std::unordered_map<int, string> label_nodes {};

    
    std::cerr << "reading graph...\n";
    
    const char* filename = "data/wikidata.rdf";

    //init variables
    CooGraph* graph_coo = (CooGraph*) malloc(sizeof(CooGraph));
    std::unordered_map<int, string> node_map {};
    std::unordered_map<string, int> node_map_reverse {};
    std::unordered_map<int, string> edge_map {};
    std::unordered_map<string, int> edge_map_reverse {};
    
    int num_nodes = 42868213;
    int num_edges = 193634832;
    
    node_map.reserve(num_nodes);
    node_map_reverse.reserve(num_nodes);
    edge_map.reserve(num_edges);
    edge_map_reverse.reserve(num_edges);

    
    //read graph and convert to CSR format
    startTime(&timer);  
    
    readGraph(filename, graph_coo, &node_map, &node_map_reverse, &edge_map, &edge_map_reverse, &label_nodes, num_nodes, num_edges, true);
    
    stopTime(&timer);
    printElapsedTime(timer, "Read graph time", CYAN);

    std::cerr << "converting to CSR...\033[0m\n";
    CsrGraph* graph = (CsrGraph*) malloc(sizeof(CsrGraph));
    startTime(&timer);

    cooToCSR(graph_coo, graph);
    
    stopTime(&timer);
    printElapsedTime(timer, "Convert to CSR time", CYAN);

    float* node_weights = (float*) malloc(graph->num_nodes * sizeof(float));
    float avg_hops = 0;

    std::cerr << "building index...\n";
    //check if index exists
    startTime(&timer);
    
    if (filename == "data/euler.txt") {
        readGraphIndex("index/euler_index.txt", node_weights, avg_hops, graph->num_nodes);
    }

    else if (access("index/wikidata_index.txt", F_OK) == 0) {
        readGraphIndex("index/wikidata_index.txt", node_weights, avg_hops, graph->num_nodes);
    }
    else {
        getVertexInformativeness(graph, graph_coo, node_weights, avg_hops);
        writeGraphIndex("index/wikidata_index.txt", node_weights, avg_hops, graph->num_nodes);
    }

    stopTime(&timer);

    printElapsedTime(timer, "build index time", CYAN);
    freeGraph(graph_coo);

    CsrGraph *graph_d = createEmptyCsrGPU(num_nodes, num_edges);

    // copy graph to device
    copyCsrGraphToDevice(graph, graph_d);

    //init result and hyperparameters
    int k = 2;
    float alpha = 0.1f;

    int* min_activations = (int*) malloc(sizeof(int) * graph->num_nodes);
    getMinActivations(node_weights, graph->num_nodes, alpha, avg_hops, min_activations);

    int* min_activations_d;
    hipMalloc((void**) &min_activations_d, sizeof(int) * graph->num_nodes);
    hipMemcpy(min_activations_d, min_activations, sizeof(int) * graph->num_nodes, hipMemcpyHostToDevice);

    while (true) {


        std::cerr << "Please enter your query (Enter 'exit' to quit):\n";
        CooGraph** results = (CooGraph**) malloc(sizeof(CooGraph*) * k);
        std::string input;

        std::getline(std::cin, input);
        if (input == "exit") {
            break;
        }

        std::vector<std::string> query {};
        std::stringstream ss(input);

        std::string token;
        while (std::getline(ss, token, ' ')) {
            std::transform(token.begin(), token.end(), token.begin(), ::tolower);
            query.push_back(token);
        }
        

        std::cerr << "getting query vertices...\n";
        //init query vertices
        std::vector<std::unordered_set<int>> keyword_nodes (query.size(), std::unordered_set<int> {});
        startTime(&timer);
        
        getQueryVertices(query, label_nodes, graph->num_nodes, keyword_nodes);

        stopTime(&timer);

        printElapsedTime(timer, "get query vertices time", CYAN);

        bool valid = true;

        for(int keyword = 0; keyword < keyword_nodes.size(); ++keyword) {
            if (keyword_nodes[keyword].size() == 0) {
                std::cerr << "keyword " << query[keyword] << " not found in graph\n";
                valid = false;
            }
        }


        if (!valid) {
            continue;
        }


        //run pks
        startTime(&timer);

        pks(graph, keyword_nodes, results, k, alpha, node_weights, min_activations, avg_hops);
            
        stopTime(&timer);

        printElapsedTime(timer, " total PKS time", GREEN);

        //print results
        for(int i = 0; i < k; ++i) {
            //reset color
            printGraph(results[i], node_map, edge_map, "result.txt");
        }
        std::cerr << "wrote result to result.txt\n";

        for(int i = 0; i < k; ++i) {
            freeGraph(results[i]);
        }

        free(results);

        results = (CooGraph**) malloc(sizeof(CooGraph*) * k);
        std::cerr << "running pks_gpu...\n";

        startTime(&timer);
        
        pks_gpu(graph, graph_d,  keyword_nodes, results, k, alpha, node_weights, min_activations, min_activations_d, avg_hops);

        stopTime(&timer);

        printElapsedTime(timer, " total PKS GPU time", GREEN);
        
        //print results
        for(int i = 0; i < k; ++i) {
            printGraph(results[i], node_map, edge_map, "result_gpu.txt");   
        }
        std::cerr << "wrote result to result_gpu.txt\n";

        for(int i = 0; i < k; ++i) {
            freeGraph(results[i]);
        }

        free(results);
        results = (CooGraph**) malloc(sizeof(CooGraph*) * k);
        std::cerr << "running pks_bitwise...\n";

        startTime(&timer);
        pks_gpu_bitwise(graph, graph_d, keyword_nodes, results, k, alpha, node_weights, min_activations, min_activations_d, avg_hops);

        stopTime(&timer);

        printElapsedTime(timer, " total PKS GPU bitwise time", GREEN);

        //print results
        for(int i = 0; i < k; ++i) {
            printGraph(results[i], node_map, edge_map, "result_bitwise.txt");   
        }

        std::cerr << "wrote result to result_bitwise.txt\n";

        for(int i = 0; i < k; ++i) {
            freeGraph(results[i]);
        }

        free(results);
    }
    free(node_weights);
    free(min_activations);
    hipFree(min_activations_d);
    cudaFreeGraph(graph_d);
    freeGraph(graph);
    return 0;
}
