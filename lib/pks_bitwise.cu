#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <unordered_set>
#include "graph.h"
#include "pks.h"
#include "util.h"
#include "index.h"
#include "timer.h"
#include "top_down.h"
#include "gpu_util.h"
#include "top_down.h"
#define INF 2147483647
#define ull unsigned long long
#define MAX_THREADS 1024

void pks_gpu_bitwise(const CsrGraph *graph, const CsrGraph* graph_d,  const std::vector<std::unordered_set<int>>& keyword_nodes, CooGraph **result, const int k, const float alpha, const float *node_weights, const int* min_activations_h, const int* min_activations_d, const float avg_hops){
    hipError_t err; Timer timer;


    int query_num = keyword_nodes.size();
    int num_nodes = graph->num_nodes;


    startTime(&timer);
    // allocate memory for graph

    int *M_d;
    unsigned long long* frontier_d, *F_identifier_d, *C_identifier_d;
    unsigned long long* keyword_nodes_d;


    //allocate memory


    int num_nodes_longlong = (num_nodes + 64 - 1 )  / 64;

    err = hipMalloc((void **)&frontier_d, (num_nodes_longlong) * sizeof(ull));
    if (err != hipSuccess){
        // Handle memory allocation error
        std::cerr << "Error allocating memory for frontier_d" << std::endl;
    }

    err = hipMalloc((void **)&F_identifier_d, num_nodes_longlong * sizeof(ull));
    if (err != hipSuccess){
        // Handle memory allocation error
        std::cerr << "Error allocating memory for F_identifier_d" << std::endl;
    }

    err = hipMalloc((void **)&C_identifier_d, num_nodes_longlong * sizeof(ull));
    if (err != hipSuccess){
        // Handle memory allocation error
        std::cerr << "Error allocating memory for C_identifier_d" << std::endl;
    }

    err = hipMalloc((void **)&keyword_nodes_d, num_nodes_longlong * sizeof(ull));
    if (err != hipSuccess){
        // Handle memory allocation error
        std::cerr << "Error allocating memory for keyword_nodes_d" << std::endl;
    }

    err = hipMalloc((void **)&M_d, sizeof(int) * num_nodes * query_num);
    if (err != hipSuccess){
        // Handle memory allocation error
        std::cerr << "Error allocating memory for M_d" << std::endl;
    }

    // Check if any memory allocation failed
    if (err != hipSuccess) {
        // Clean up allocated memory
        hipFree(M_d);
        hipFree(keyword_nodes_d);
        hipFree(C_identifier_d);
        hipFree(F_identifier_d);
        hipFree(frontier_d);
        // Return or throw an appropriate error
        return;
    }

    err = hipMemsetAsync(frontier_d, 0, num_nodes_longlong * sizeof(ull));
    if (err != hipSuccess){
        // Handle memory initialization error
        std::cerr << "Error initializing frontier_d" << std::endl;
    }

    err = hipMemsetAsync(C_identifier_d, 0, num_nodes_longlong * sizeof(ull));
    if (err != hipSuccess){
        // Handle memory initialization error
        std::cerr << "Error initializing C_identifier_d" << std::endl;
    }


    init_M_keywords_bitwise(keyword_nodes_d, M_d, num_nodes, query_num, keyword_nodes);

    err = hipGetLastError();
    if (err != hipSuccess){
        // Handle memory initialization error
        std::cerr << "cuda error: " << hipGetErrorString(err) << "\n";
    }

    hipMemcpy(F_identifier_d, keyword_nodes_d, num_nodes_longlong * sizeof(ull) , hipMemcpyDeviceToDevice);
    
    int BFS_level = 0;
    bool terminate = false;
    std::cerr << "starting gpu expansion\n";

    stopTime(&timer);
    printElapsedTime(timer, "gpu copy and init time: ");

    startTime(&timer);

    while (!terminate){
        enqueue_frontier_bitwise(num_nodes, F_identifier_d, frontier_d);

        // TODO
        expand_bitwise(graph_d, frontier_d, F_identifier_d, M_d, C_identifier_d, min_activations_d, BFS_level, alpha, avg_hops, keyword_nodes_d, query_num, num_nodes);

        dequeue_frontier_bitwise(frontier_d, num_nodes);

        identify_central_bitwise(num_nodes, C_identifier_d, F_identifier_d, M_d, query_num);

        BFS_level++;

        terminate = check_terminate_bitwise(C_identifier_d, num_nodes, k);
    }
    stopTime(&timer);

    printElapsedTime(timer, "gpu expansion time: ");
    // copy back to host

    startTime(&timer);

    int *M = (int *)malloc(sizeof(int) * num_nodes * query_num);
    hipMemcpy(M, M_d, sizeof(int) * num_nodes * query_num, hipMemcpyDeviceToHost);

    ull *C_identifier_h = (ull* ) malloc(num_nodes_longlong * sizeof(ull));
    hipMemcpy(C_identifier_h, C_identifier_d, num_nodes_longlong * sizeof(ull), hipMemcpyDeviceToHost);

    // free memory
    hipFree(frontier_d);
    hipFree(F_identifier_d);
    hipFree(C_identifier_d);
    hipFree(keyword_nodes_d);
    hipFree(M_d);

    std::cerr << "start topdown construct\n";

    topdown_construct(graph, result, C_identifier_h, M, query_num, k, keyword_nodes, min_activations_h, node_weights);

    std::cerr << "end topdown construct\n";

    // free memory
    free(M);
    free(C_identifier_h);
}

__global__ void expand_kernel_bitwise(const CsrGraph* graph, const ull* frontier, ull* F_identifier, int* M, ull* C_indentifier, const int* min_activations, int bfs_level, float alpha, float avg_hops, const ull* keyword_nodes, int query_num) {
    int node = blockIdx.x * blockDim.x + threadIdx.x;

    if (node < graph->num_nodes) {
        ull node_bit = (1ULL << (node % 64));
        if (frontier[node / 64] & node_bit) {
            if (!(C_indentifier[node / 64] & node_bit)) {
                int min_activation_level = min_activations[node];

                if (min_activation_level > bfs_level) {
                    atomicOr(&F_identifier[node / 64], node_bit);
                    return;
                }

                for(int bfs_instance = 0; bfs_instance < query_num; ++bfs_instance) {
                    
                    int hitting_level = M[node * query_num + bfs_instance];

                    if (hitting_level > bfs_level) {
                        continue;
                    }

                    for(int neighbor = graph->row_offsets[node]; neighbor < graph->row_offsets[node + 1]; ++neighbor) {
                        int neighbor_id = graph->col_indices[neighbor];

                        int neighbor_hitting_level = M[neighbor_id * query_num + bfs_instance];

                        if (neighbor_hitting_level != INF)
                            continue;

                        if (! (keyword_nodes[neighbor_id / 64] & node_bit ) ) {
                            int neighbor_activation_level = min_activations[neighbor_id];
                            if (neighbor_activation_level > bfs_level + 1) {
                                atomicOr(&F_identifier[neighbor_id / 64], node_bit);
                                continue;
                            }
                        }
                        M[neighbor_id * query_num + bfs_instance] = bfs_level + 1;
                        
                        atomicOr(&F_identifier[neighbor_id / 64], (1ULL << (neighbor_id % 64)));
                    }
                }
            }
        }
    }
}


void expand_bitwise(const CsrGraph* graph_d, const ull* frontier_d, ull* F_identifier_d, int* M_d, ull* C_identifier_d, const int* min_activations_d, int bfs_level, float alpha, float avg_hops, const ull* keyword_nodes_d, int query_num, int num_nodes) {

    int num_threads = MAX_THREADS; 
    int num_blocks = (num_nodes + num_threads - 1) / num_threads;

    expand_kernel_bitwise<<<num_blocks, num_threads>>>(graph_d, frontier_d, F_identifier_d, M_d, C_identifier_d, min_activations_d, bfs_level, alpha, avg_hops, keyword_nodes_d, query_num);

    hipDeviceSynchronize();
}
